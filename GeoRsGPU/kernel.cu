#include "hip/hip_runtime.h"
#include <chrono>
#include <fstream>
#include <functional>
#include <iostream>
#include <iomanip>
#include <string>

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "gdal_priv.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

void timeIt(std::string operationName, std::function<void()> func)
{
	auto startTime = std::chrono::high_resolution_clock::now();
	func();
	auto endTime = std::chrono::high_resolution_clock::now();


	int totalMilliseconds = std::chrono::
		duration_cast<std::chrono::milliseconds>(endTime - startTime).count();

	std::cout << "[" << operationName << "]: " << totalMilliseconds << "ms" << std::endl;
}

void executeCuda(std::function<hipError_t()> cudaFunc)
{
	hipError_t code = cudaFunc();
	if (code != hipSuccess)
	{
		fprintf(stderr, "CUDA ERROR: %d %s\n", code, hipGetErrorString(code));
		exit(code);
	}
}

void checkCuda()
{
	hipError_t code = hipGetLastError();
	if (code != hipSuccess)
	{
		fprintf(stderr, "CUDA ERROR: %d %s\n", code, hipGetErrorString(code));
		exit(code);
	}
}
void exitProgram(const char* const message)
{
	fprintf(stderr, "\n");
	fprintf(stderr, message);
	fprintf(stderr, "\n");
	exit(1);
}

void readInputFromFile(
	// Input:
	const char* const filePath,
	// Output:
	float*& inputData, double geoTransform[], int& nLineSize, int& nNumberOfLines,
	GDALDriver*& poDriver)
{
	GDALAllRegister();

	GDALDataset* poDataset = (GDALDataset*)GDALOpen(filePath, GA_ReadOnly);
	if (poDataset == NULL)
	{
		exitProgram("ERROR: Failed to open GDAL dataset.");
	}

	poDriver = GetGDALDriverManager()->GetDriverByName(poDataset->GetDriverName());
	if (poDriver == NULL)
	{
		exitProgram("ERROR: Failed to identify the GDAL driver.");
	}

	// In the particular, but common, case of a "north up" image without any rotation or shearing, 
	// the georeferencing transform takes the following form :
	// 0 -> top left x
	// 1 -> w-e pixel resolution
	// 2 -> 0
	// 3 -> top left y
	// 4 -> 0
	// 5 -> n-s pixel resolution (negative value)
	poDataset->GetGeoTransform(geoTransform);

	// We assume that the file has only one raster band
	// (we always use the first one).
	GDALRasterBand* poRasterBand = poDataset->GetRasterBand(1);
	nLineSize = poRasterBand->GetXSize();
	nNumberOfLines = poRasterBand->GetYSize();

	// Allocate pinned host memory (faster CUDA transfer performance)
	executeCuda([&]() { return hipHostMalloc((void**)&inputData, nLineSize * nNumberOfLines * sizeof(float)); });

	// Read data from file
	poRasterBand->RasterIO(GDALRWFlag::GF_Read, 0, 0, nLineSize, nNumberOfLines, inputData, nLineSize, nNumberOfLines, GDALDataType::GDT_Float32, 0, 0);

	GDALClose(poDataset);
}


int main()
{
	/*** Load input data from DEM file => inputData, geoTransform, nLineSize, nNumberOfLines, fCellSizeX, fCellSizeY, poDriver ***/
	double geoTransform[6];
	int nLineSize;
	int nNumberOfLines;
	float* inputData;
	GDALDriver *poDriver;

	timeIt("Load Data", [&]() {
		readInputFromFile(
			// input:
			"d:\\GeoGPUTeste\\Data\\dem9112.tif", // input file path
					 // output:
			inputData, geoTransform, nLineSize, nNumberOfLines, poDriver);
	});

	float fCellSizeX = (float)geoTransform[1];
	float fCellSizeY = -(float)geoTransform[5];

	std::cout << "Cell size: " << fCellSizeX << ", " << fCellSizeY << "; Raster size: " << nNumberOfLines << " rows x " << nLineSize << " columns" << std::endl;

    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
