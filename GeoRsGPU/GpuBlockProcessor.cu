#include "hip/hip_runtime.h"
// MIT License
// 
// Copyright(c) 2017 Cristian Ionita, Ionut Sandric, Marian Dardala, Titus Felix Furtuna
// 
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files(the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and / or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions :
// 
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include <stdexcept>

#include "GpuBlockProcessor.cuh"
#include "DemKernels.cuh"
#include "LocalStatisticsKernels.cuh"
#include "TrigonometricKernels.cuh"
#include "MathKernels.cuh"

using namespace GeoRsGpu;

const int MAX_ERROR_MESSAGE_LEN = 300;

GpuBlockProcessor::GpuBlockProcessor(
	CommandLineParser& parser,
	int maxBlockHeight, int maxBlockWidth,
	float cellSizeX, float cellSizeY)
	: m_commandLineParser(parser)
{
	m_command = m_commandLineParser.getCommand();
	m_maxBlockHeight = maxBlockHeight;
	m_maxBlockWidth = maxBlockWidth;

	m_cellSizeX = cellSizeX;
	m_cellSizeY = cellSizeY;

	/*** Allocate host and device memory for two blocks (in / out) ***/
	executeCuda([&]() { return hipHostMalloc((void**)&m_in,
		maxBlockHeight * maxBlockWidth * sizeof(float)); });
	checkCuda();

	executeCuda([&]() { return hipHostMalloc((void**)&m_out,
		maxBlockHeight * maxBlockWidth * sizeof(float)); });
	checkCuda();

	executeCuda([&]() { return hipMalloc((void**)&m_devIn,
		maxBlockHeight * maxBlockWidth * sizeof(float)); });
	checkCuda();

	executeCuda([&]() { return hipMalloc((void**)&m_devOut,
		maxBlockHeight * maxBlockWidth * sizeof(float)); });
	checkCuda();
}

GpuBlockProcessor::~GpuBlockProcessor()
{
	executeCuda([&]() { return hipHostFree((void**)m_in); });
	executeCuda([&]() { return hipHostFree((void**)m_out); });
	executeCuda([&]() { return hipFree((void**)m_devIn); });
	executeCuda([&]() { return hipFree((void**)m_devOut); });
	executeCuda([&]() { return hipDeviceSynchronize(); });
}

template <typename EQ>
__global__ void gpuKernel(
	const float * const __restrict input, float * const __restrict output,
	const int height, const int width,
	const int heightOut, const int widthOut,
	const int deltaRow, const int deltaCol,
	const float cellSizeX, const float cellSizeY)
{
	int rowIndex = blockIdx.y * blockDim.y + threadIdx.y;
	int colIndex = blockIdx.x * blockDim.x + threadIdx.x;

	int colIndexOut = colIndex + deltaCol;
	int rowIndexOut = rowIndex + deltaRow;

	if (colIndexOut < 0 || colIndexOut >= widthOut
		|| rowIndexOut < 0 || rowIndexOut >= heightOut)
	{
		// We don't have any output value
		return;
	}

	float& outputElem = output[rowIndexOut * widthOut + colIndexOut];

	if (colIndex > 0 && colIndex < width - 1
		&& rowIndex > 0 && rowIndex < height - 1)
	{
		// We have everything we need
		float a = input[width * (rowIndex - 1) + colIndex - 1];
		float b = input[width * (rowIndex - 1) + colIndex];
		float c = input[width * (rowIndex - 1) + colIndex + 1];

		float d = input[width * rowIndex + colIndex - 1];
		float e = input[width * rowIndex + colIndex];
		float f = input[width * rowIndex + colIndex + 1];

		float g = input[width * (rowIndex + 1) + colIndex - 1];
		float h = input[width * (rowIndex + 1) + colIndex];
		float i = input[width * (rowIndex + 1) + colIndex + 1];

		outputElem = EQ()(a, b, c, d, e, f, g, h, i, cellSizeX, cellSizeY);
	}
	else if (
		colIndex == 0 || rowIndex == 0 ||
		colIndex == width - 1 || rowIndex == height - 1)
	{
		// We are on the edge - we don't have all surrounding values
		outputElem = 0;
	}
}
template <typename EQ>
__global__ void gpuKernel_NXM(
	const float * const __restrict input, float * const __restrict output,
	const int height, const int width,
	const int heightOut, const int widthOut,
	const int deltaRow, const int deltaCol,
	const float cellSizeX, const float cellSizeY)
{
	int rowIndex = blockIdx.y * blockDim.y + threadIdx.y;
	int colIndex = blockIdx.x * blockDim.x + threadIdx.x;

	int colIndexOut = colIndex + deltaCol;
	int rowIndexOut = rowIndex + deltaRow;

	if (colIndexOut < 0 || colIndexOut >= widthOut
		|| rowIndexOut < 0 || rowIndexOut >= heightOut)
	{
		// We don't have any output value
		return;
	}

	float& outputElem = output[rowIndexOut * widthOut + colIndexOut];

	if (colIndex > 0 && colIndex < width - 1
		&& rowIndex > 0 && rowIndex < height - 1)
	{
		// We have everything we need
		float a = input[width * (rowIndex - 1) + colIndex - 1];
		float b = input[width * (rowIndex - 1) + colIndex];
		float c = input[width * (rowIndex - 1) + colIndex + 1];

		float d = input[width * rowIndex + colIndex - 1];
		float e = input[width * rowIndex + colIndex];
		float f = input[width * rowIndex + colIndex + 1];

		float g = input[width * (rowIndex + 1) + colIndex - 1];
		float h = input[width * (rowIndex + 1) + colIndex];
		float i = input[width * (rowIndex + 1) + colIndex + 1];

		outputElem = EQ()(a, b, c, d, e, f, g, h, i, cellSizeX, cellSizeY);
	}
	else if (
		colIndex == 0 || rowIndex == 0 ||
		colIndex == width - 1 || rowIndex == height - 1)
	{
		// We are on the edge - we don't have all surrounding values
		outputElem = 0;
	}
}

void GpuBlockProcessor::processBlock(BlockRect rectIn, BlockRect rectOut)
{
	size_t blockSizeBytes = rectIn.getWidth() * rectIn.getHeight() * sizeof(float);
	executeCuda([&]() { return hipMemcpy(
		m_devIn, m_in, blockSizeBytes, hipMemcpyHostToDevice); });
	executeCuda([&]() { return hipDeviceSynchronize(); });

	dim3 grid;
	dim3 block(16, 16);
	grid.x = rectIn.getWidth() / block.x + (rectIn.getWidth() % block.x == 0 ? 0 : 1);
	grid.y = rectIn.getHeight() / block.y + (rectIn.getHeight() % block.y == 0 ? 0 : 1);

	// input and output sizes (different for edge cases)
	int inH = rectIn.getHeight(), inW = rectIn.getWidth();
	int outH = rectOut.getHeight(), outW = rectOut.getWidth();

	// delta row and column - specify how the output block
	// is positioned against the input block (non zero for edge cases)
	int dR = rectIn.getRowStart() - rectOut.getRowStart();
	int dC = rectIn.getColStart() - rectOut.getColStart();

#define KERNEL_PARAMS m_devIn, m_devOut, inH, inW, outH, outW, dR, dC, m_cellSizeX, m_cellSizeY

	switch (m_command)
	{
	case RasterCommand::Slope:
	{
		bool useBurruogh = true; // use Burruogh by default
		if (m_commandLineParser.parameterExists("Alg"))
		{
			if (m_commandLineParser.getStringParameter("Alg") == "Zvn")
			{
				useBurruogh = false;
			}
			else if (m_commandLineParser.getStringParameter("Alg") == "Brr")
			{
				useBurruogh = true;
			}
			else
			{
				throw std::runtime_error("Invalid slope algorithm.");
			}
		}
		if (useBurruogh)
		{
			gpuKernel<KernelSlopeBurruogh> << <grid, block >> > (KERNEL_PARAMS);
		}
		else
		{
			gpuKernel<KernelSlopeZevenbergen> << <grid, block >> > (KERNEL_PARAMS);
		}
	}
	break;

	case RasterCommand::Hillshade:
		gpuKernel<KernelHillshade> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::Aspect:
		gpuKernel<KernelAspect> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::TotalCurvature:
		gpuKernel<KernelTotalCurvature> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::PlanCurvature:
		gpuKernel<KernelPlanCurvature> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::ProfileCurvature:
		gpuKernel<KernelProfileCurvature> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::TopographicPositionIndex:
	{
		bool useEP = true;
		bool useDIF = false;
		bool useDEV = false;
		if (m_commandLineParser.parameterExists("Alg"))
		{
			if (m_commandLineParser.getStringParameter("Alg") == "EP")
			{
				useEP = true;
			}
			else if (m_commandLineParser.getStringParameter("Alg") == "DIF")
			{
				useDIF = true;
			}
			else if (m_commandLineParser.getStringParameter("Alg") == "DEV")
			{
				useDEV = true;
			}
			else
			{
				throw std::runtime_error("Invalid topographic postion index");
			}
		}
		if (useEP)
		{
			gpuKernel<KernelTopographicPositionIndex_EP> << <grid, block >> > (KERNEL_PARAMS);
		}
		else if (useDIF)
		{
			gpuKernel<KernelTopographicPositionIndex_DIF> << <grid, block >> > (KERNEL_PARAMS);
		}
		else if (useDEV)
		{
			gpuKernel<KernelTopographicPositionIndex_DEV> << <grid, block >> > (KERNEL_PARAMS);
		}
	}
	break;


	case RasterCommand::Minority:
		gpuKernel<KernelMinority> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::Majority:
		gpuKernel<KernelMajority> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::Mean:
		gpuKernel<KernelMean> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::Median:
		gpuKernel<KernelMedian> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::Maximum:
		gpuKernel<KernelMaximum> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::Minimum:
		gpuKernel<KernelMinimum> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::StandardDeviation:
		gpuKernel<KernelStandardDeviation> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::Range:
		gpuKernel<KernelRange> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::Variety:
		gpuKernel<KernelVariety> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::Percentile:
		gpuKernel<KernelPercentile> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::DiffFromMean:
		gpuKernel<KernelDiffFromMean> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::StDevFromMean:
		gpuKernel<KernelDevFromMean> << <grid, block >> > (KERNEL_PARAMS);
		break;


		//Trigonometric kernels
	case RasterCommand::ATan:
		gpuKernel<KernelATAN> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::ATanH:
		gpuKernel<KernelATANH> << <grid, block >> > (KERNEL_PARAMS);
		break;

		//case RasterCommand::ATan2:
		//	gpuKernel<KernelATAN2> << <grid, block >> > (KERNEL_PARAMS);
		//	break;

	case RasterCommand::ACos:
		gpuKernel<KernelACos> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::ACosH:
		gpuKernel<KernelACosH> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::ASin:
		gpuKernel<KernelASin> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::ASinH:
		gpuKernel<KernelASinH> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::Tan:
		gpuKernel<KernelTAN> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::TanH:
		gpuKernel<KernelTANH> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::Cos:
		gpuKernel<KernelCos> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::CosH:
		gpuKernel<KernelCosH> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::Sin:
		gpuKernel<KernelSin> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::SinH:
		gpuKernel<KernelSinH> << <grid, block >> > (KERNEL_PARAMS);
		break;


		//Math functions
	case RasterCommand::Abs:
		gpuKernel<KernelAbs> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::RoundUp:
		gpuKernel<KernelCeil> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::RoundDown:
		gpuKernel<KernelFloor> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::Exp10f:
		gpuKernel<KernelExp10f> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::Exp2f:
		gpuKernel<KernelExp2f> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::Lnf:
		gpuKernel<KernelLogf> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::Log10f:
		gpuKernel<KernelLog10f> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::Log2f:
		gpuKernel<KernelLog2f> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::Negate:
		gpuKernel<KernelNegate> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::Power:
		gpuKernel<KernelPower> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::Square:
		gpuKernel<KernelSquare> << <grid, block >> > (KERNEL_PARAMS);
		break;

	case RasterCommand::SqRoot:
		gpuKernel<KernelSquareRoot> << <grid, block >> > (KERNEL_PARAMS);
		break;



	default:
		char buffer[MAX_ERROR_MESSAGE_LEN];
		snprintf(buffer, sizeof(buffer),
			"Command #%d is not supported.", m_command);
		throw std::runtime_error(buffer);
	}

	executeCuda([&]() { return hipDeviceSynchronize(); });

	executeCuda([&]() { return hipMemcpy(
		m_out, m_devOut, blockSizeBytes, hipMemcpyDeviceToHost); });
	executeCuda([&]() { return hipDeviceSynchronize(); });
}

void GpuBlockProcessor::executeCuda(std::function<hipError_t()> cudaFunc)
{
	hipError_t code = cudaFunc();
	if (code != hipSuccess)
	{
		char buffer[MAX_ERROR_MESSAGE_LEN];
		snprintf(buffer, sizeof(buffer),
			"CUDA #%d - %s", code, hipGetErrorString(code));
		throw std::runtime_error(buffer);
	}
}

void GpuBlockProcessor::checkCuda()
{
	hipError_t code = hipGetLastError();
	if (code != hipSuccess)
	{
		char buffer[MAX_ERROR_MESSAGE_LEN];
		snprintf(buffer, sizeof(buffer),
			"CUDA #%d - %s", code, hipGetErrorString(code));
		throw std::runtime_error(buffer);
	}
}

void GpuBlockProcessor::startCuda()
{
	/*** Init CUDA ***/
	// Choose which GPU to run on, change this on a multi-GPU system.
	executeCuda([]() { return hipSetDevice(0); });
	checkCuda();

	executeCuda([]() { return hipDeviceSetCacheConfig(hipFuncCachePreferL1); });
	checkCuda();
}

void GpuBlockProcessor::stopCuda()
{
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	executeCuda([]() { return hipDeviceReset(); });
}
